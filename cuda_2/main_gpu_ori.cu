#include <iostream>
#include <cstdlib>
#include <sys/time.h>
#include <hip/hip_runtime.h>

using namespace std;

__global__
void vecMultipleKernel(float* A_d, float* B_d, float* C_d, int n)
{
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < n) C_d[i] = A_d[i] + B_d[i];
}

int main(int argc, char *argv[]) {

    int n = atoi(argv[1]);
    cout << n << endl;

    size_t size = n * sizeof(float);

    // host memery
    float **a = (float **)malloc(n * sizeof(float*));
    float **b = (float **)malloc(n * sizeof(float*));
    float **c = (float **)malloc(n * sizeof(float*));

    for (int i = 0; i < n; i++) {
        a[i] = (float *)malloc(n * sizeof(float));
        b[i] = (float *)malloc(n * sizeof(float));
        c[i] = (float *)malloc(n * sizeof(float));

        for (int j = 0; j < n; j++) {
            a[i][j] = rand() / double(RAND_MAX);
            b[i][j] = rand() / double(RAND_MAX);
        }
    }


    float *da = NULL;
    float *db = NULL;
    float *dc = NULL;

    hipMalloc((void **)&da, size * size);
    hipMalloc((void **)&db, size * size);
    hipMalloc((void **)&dc, size * size);

    hipMemcpy(da,a,size,hipMemcpyHostToDevice);
    hipMemcpy(db,b,size,hipMemcpyHostToDevice);
    hipMemcpy(dc,c,size,hipMemcpyHostToDevice);

    struct timeval t1, t2;

    int threadPerBlock = 256;
    int blockPerGrid = (n + threadPerBlock - 1)/threadPerBlock;
    printf("threadPerBlock: %d \nblockPerGrid: %d \n",threadPerBlock,blockPerGrid);

    gettimeofday(&t1, NULL);

    vecMultipleKernel <<< blockPerGrid, threadPerBlock >>> (da, db, dc, n);

    gettimeofday(&t2, NULL);

    hipMemcpy(c,dc,size,hipMemcpyDeviceToHost);

    //for (int i = 0; i < 10; i++) 
    //    cout << vecA[i] << " " << vecB[i] << " " << vecC[i] << endl;
    double timeuse = (t2.tv_sec - t1.tv_sec) + (double)(t2.tv_usec - t1.tv_usec)/1000000.0;
    cout << timeuse << endl;

    hipFree(da);
    hipFree(db);
    hipFree(dc);

    free(a);
    free(b);
    free(c);
    return 0;
}
