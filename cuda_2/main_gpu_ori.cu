#include <iostream>
#include <cstdlib>
#include <sys/time.h>
#include <hip/hip_runtime.h>

using namespace std;

__global__
void vecMultiplicationKernel(float* A_d, float* B_d, float* C_d, int n)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < n && y < n) {
        float sum = 0;
        for (int k = 0; k < n; k++) {
            sum += A_d[y * n + k] * B_d[k * n + x];
        }
        C_d[y * n + x] = sum;
    }
}

void visit_2d_array(float *data, int n) {
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            printf("%f ", data[i * n + j]);
        }
        printf("\n");
    }
    printf("\n");
}

void initial_matrix(float *data, int n) {
    for (int i = 0; i < n; i++) {
        data[i] = rand() / double(RAND_MAX);
    }
}

int main(int argc, char *argv[]) {

    int n = atoi(argv[1]);
    cout << n << endl;

    size_t size = n * n * sizeof(float);

    // host memery
    float *a = (float *)malloc(size);
    float *b = (float *)malloc(size);
    float *c = (float *)malloc(size);

    initial_matrix(a, n * n);
    initial_matrix(b, n * n);


    float *da = NULL;
    float *db = NULL;
    float *dc = NULL;

    hipMalloc((void **)&da, size);
    hipMalloc((void **)&db, size);
    hipMalloc((void **)&dc, size);

    hipMemcpy(da,a,size,hipMemcpyHostToDevice);
    hipMemcpy(db,b,size,hipMemcpyHostToDevice);
    hipMemcpy(dc,c,size,hipMemcpyHostToDevice);

    struct timeval t1, t2;

    // int threadPerBlock = 256;
    // int blockPerGrid = (n + threadPerBlock - 1)/threadPerBlock;
    // printf("threadPerBlock: %d \nblockPerGrid: %d \n",threadPerBlock,blockPerGrid);

    int dimx = 5, dimy = 5;
    dim3 block(dimx, dimy);
    dim3 grid((n + block.x - 1) / block.x, (n + block.y - 1)/block.y);

    gettimeofday(&t1, NULL);

    vecMultiplicationKernel <<< block, grid >>> (da, db, dc, n);

    gettimeofday(&t2, NULL);

    hipMemcpy(c,dc,size,hipMemcpyDeviceToHost);

    //for (int i = 0; i < 10; i++) 
    //    cout << vecA[i] << " " << vecB[i] << " " << vecC[i] << endl;
    double timeuse = (t2.tv_sec - t1.tv_sec) + (double)(t2.tv_usec - t1.tv_usec)/1000000.0;
    cout << timeuse << endl;

    hipFree(da);
    hipFree(db);
    hipFree(dc);

    // visit_2d_array(a, n);
    // visit_2d_array(b, n);
    // visit_2d_array(c, n);

    free(a);
    free(b);
    free(c);
    return 0;
}
